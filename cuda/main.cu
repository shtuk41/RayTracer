#include "hip/hip_runtime.h"
//https://raytracing.github.io/books/RayTracingInOneWeekend.html#surfacenormalsandmultipleobjects/somenewc++features
//https://developer.nvidia.com/blog/accelerated-ray-tracing-cuda/
//http://richiesams.blogspot.com/2015/03/creating-randomness-and-acummulating.html

#include <algorithm>
#include <fstream>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <ctime>

#include <opencv2/opencv.hpp>
#include "vec3.h"
#include "ray.h"
#include "color.h"
#include "sphere.h"
#include "hitable_list.h"
#include "camera.h"
#include "lambertian.h"
#include "metal.h"
#include "dielectric.h"

#define CHECK(call) \
{                                                       \
    const hipError_t error = call;                     \
    if (error != hipSuccess)                           \
    {                                                   \
        printf("Error: %s,%d, ", __FILE__, __LINE__);   \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error));\
        hipDeviceReset();\
        exit(1);\
    }\
}

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ color ray_color(const ray &r, hitable **world, hiprandState *local_rand_state)
{
   ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0,1.0,1.0);
    for(int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if(rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return vec3(0.0,0.0,0.0);
            }
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f*(unit_direction.y() + 1.0f);
            vec3 c = (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0,0.0,0.0); // exceeded recursion
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state)
{
   int i = threadIdx.x + blockIdx.x * blockDim.x;
   int j = threadIdx.y + blockIdx.y * blockDim.y;
   
   if((i >= max_x) || (j >= max_y)) 
       return;
   
   int pixel_index = j*max_x + i;
   //Each thread gets same seed, a different sequence number, no offset
   //hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
   hiprand_init(pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns, camera **cam, hitable **world, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0,0,0);
    
    for (int s = 0; s < ns; s++)
    {
    
        float u = float(i + hiprand_uniform(&local_rand_state)) / (max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / (max_y);
        ray r = (*cam)->get_ray(u,v);
        col += ray_color(r, world, &local_rand_state);
    }
    
    fb[pixel_index] = (col/float(ns)).squareroot();
}

__global__ void create_world(hitable **d_list, hitable **d_world, camera **cam)
{
    if (threadIdx.x == 0 && blockIdx.x == 0) 
    {
        lambertian *material_ground = new lambertian(color(0.8f, 0.8f, 0.0f));
        lambertian *material_center = new lambertian(color(0.1f, 0.2f, 0.5f));
        
        //metal* material_left   = new metal(color(0.8f, 0.8f, 0.8f), 0.2f);
        
        dielectric* material_left   = new dielectric(1.5f);
        metal* material_right  = new metal(color(0.8f, 0.6f, 0.2f), 0.0f);
        
        *(d_list)   = new sphere(point3( 0.0, -100.5, -1.0), 100.0, material_ground);
        *(d_list+1) = new sphere(point3( 0.0,    0.0, -1.0),   0.5, material_center);
        *(d_list+2) = new sphere(point3( -1.0,    0.0, -1.0),   0.5, material_left);
        *(d_list+3) = new sphere(point3( 1.0,    0.0, -1.0),   0.5, material_right);
        *d_world    = new hitable_list(d_list,4);
        *cam        = new camera();
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera) {
    delete *(d_list);
    delete *(d_list+1);
    delete *(d_list+2);
    delete *(d_list+3);
    delete *d_world;
    delete *d_camera;
}

int main(int argc, char **argv)
{  
    //image
    camera **d_camera;
    CHECK(hipMalloc((void**)&d_camera, sizeof(camera *)));
    
    const float aspect_ratio = 16.0f / 9.0f;
    const int image_width = 2000;
    const int image_height = static_cast<int>(image_width / aspect_ratio);
    
 
    
    hitable **d_list;
    CHECK(hipMalloc((void**)&d_list, 2 * sizeof(hitable *)));
    hitable **d_world;
    CHECK(hipMalloc((void **)&d_world, sizeof(hitable *)));
    create_world<<<1,1>>>(d_list,d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    
    int nx = image_width;
    int ny = image_height;
    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(vec3);

    // allocate FB
    vec3 *fb;
    CHECK(hipMallocManaged((void **)&fb, fb_size));
     
    int tx = 32;
    int ty = 32;
    
    // Render our buffer
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    
     hiprandState *d_rand_state;
    CHECK(hipMalloc((void **)&d_rand_state, num_pixels * sizeof(hiprandState)));
    
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    
    int number_samples_per_pixel = 500;
    
    
    render<<<blocks, threads>>>(fb, nx, ny, number_samples_per_pixel, d_camera, d_world, d_rand_state);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());
    
    // Output FB as Image
    std::ofstream imageFile;
    
    imageFile.open("imageFile.ppm");
    
    if (!imageFile.is_open())
    {
        std::cout << "unable to open file" << std::endl;
        return -1;
    }
    
    imageFile << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--) 
    {
        for (int i = 0; i < nx; i++) 
        {
       
            size_t pixel_index = j*nx + i;
                        
            write_color(imageFile, fb[pixel_index]);
            
            //imageFile << ir << " " << ig << " " << ib << "\n";
        }
    }
    
    imageFile.close();
    
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world,d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(fb));
    
     // useful for cuda-memcheck --leak-check full
    hipDeviceReset();
    
    return 0;

}




        
